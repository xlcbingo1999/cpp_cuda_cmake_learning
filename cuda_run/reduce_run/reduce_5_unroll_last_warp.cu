// 由于在一个warp中，所有的线程都是执行一样的逻辑，且他们是强同步的，不需要使用__syncthreads()进行强制同步也是可以的
// 第二，如果在一个wrap中，则不需要再进行一次if(threadIdx.x < i)的判断也可以，直接让强同步的所有thread都执行一样的操作
// 但是，有个需要关注的点是，当程序在并行执行的时候，编译器可能会直接从寄存器中读取值，而不是直接从shared_memory中读取，会存在一致性的问题，因此需要 关键字，避免编译器进行优化

#include <stdio.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256

// 需要用volatile关闭编译器优化
__device__ void unroll_last_warp(volatile __shared__ float *shared_d_input) {
    if (threadIdx.x < 32) {
        shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + 16];
        shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + 8];
        shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + 4];
        shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + 2];
        shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + 1];
    }
}

__global__ void reduce_2(float *d_input, float *d_output) {
    // 首先, 需要让block中的每个元素都拷贝到shared memory上
    float *d_input_begin = d_input + blockDim.x * blockIdx.x * 2; // 现在一次读取都是跨两个block的
    __shared__ float shared_d_input[THREAD_PER_BLOCK]; // 需要单独申请一段shared memory空间，大小也不需要变化
    shared_d_input[threadIdx.x] = d_input_begin[threadIdx.x] + d_input_begin[threadIdx.x + blockDim.x];
    __syncthreads(); // 因为拷贝到shared memory也是并行化操作，所以需要同步thread
    
    // 避免让两个thread同时处理一个wrap中的bank，因此每次都是跨很大的距离进行求和
    for (int i = blockDim.x / 2; i > 32; i = i / 2) {
        // 这个操作就是让前面一半的thread都进入if逻辑，后面一半的thread不进入if逻辑
        if (threadIdx.x < i) {
            shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + i];
        }
        __syncthreads();
    }

    unroll_last_warp(shared_d_input);    

    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = shared_d_input[0];
    }
}

bool check(float *output, float *result, int block_num) {
    for (int i = 0; i < block_num; i++) {
        if (abs(output[i] - result[i]) > 1e-2) {
            printf("index %d wrong\n", i);
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 2 * 1024 * 1024;
    int block_num = N / THREAD_PER_BLOCK / 2; // 只需要一半的数据被block覆盖即可
    float *input = (float*)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void**)&d_input, N * sizeof(float));

    float *output = (float*)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void**)&d_output, block_num * sizeof(float));
    

    float *result = (float*)malloc(block_num * sizeof(float));
    for (int i = 0; i < block_num; i++) {
        float res = 0.0;
        for (int j = 0; j < THREAD_PER_BLOCK * 2; j++) {
            res += input[i * THREAD_PER_BLOCK * 2 + j];
        }
        result[i] = res;
    }

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1); // 一个block中的thread数量还是不需要变的，只需要让一半的数据被block覆盖即可

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    reduce_2<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, result, block_num)) {
        printf("all ok\n");
    } else {
        printf("some wrong\n");
    }

    free(result);
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}