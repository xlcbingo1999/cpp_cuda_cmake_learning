// 本代码避免了让warp中的同一个thread同时访问一个warp中的数据，避免了warp的bank conflict
// 核心就是让GPU的一个wrap内的线程的访存不需要fifo排队等待，而是直接就可以一次性读完数据

#include <stdio.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256

__global__ void reduce_2(float *d_input, float *d_output) {
    // 首先, 需要让block中的每个元素都拷贝到shared memory上
    float *d_input_begin = d_input + blockDim.x * blockIdx.x;
    __shared__ float shared_d_input[THREAD_PER_BLOCK]; // 需要单独申请一段shared memory空间
    shared_d_input[threadIdx.x] = d_input_begin[threadIdx.x];
    __syncthreads(); // 因为拷贝到shared memory也是并行化操作，所以需要同步thread
    
    // 避免让两个thread同时处理一个wrap中的bank，因此每次都是跨很大的距离进行求和
    for (int i = blockDim.x / 2; i > 0; i = i / 2) {
        // 这个操作就是让前面一半的thread都进入if逻辑，后面一半的thread不进入if逻辑
        if (threadIdx.x < i) {
            shared_d_input[threadIdx.x] += shared_d_input[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = shared_d_input[0];
    }
}

bool check(float *output, float *result, int block_num) {
    for (int i = 0; i < block_num; i++) {
        if (abs(output[i] - result[i]) > 1e-2) {
            printf("index %d wrong\n", i);
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 2 * 1024 * 1024;
    int block_num = N / THREAD_PER_BLOCK;
    float *input = (float*)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void**)&d_input, N * sizeof(float));

    float *output = (float*)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void**)&d_output, block_num * sizeof(float));
    

    float *result = (float*)malloc(block_num * sizeof(float));
    for (int i = 0; i < block_num; i++) {
        float res = 0.0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++) {
            res += input[i * THREAD_PER_BLOCK + j];
        }
        result[i] = res;
    }

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    reduce_2<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, result, block_num)) {
        printf("all ok\n");
    } else {
        printf("some wrong\n");
    }

    free(result);
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}